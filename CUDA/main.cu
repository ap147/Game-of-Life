// Amarjot Singh Parmar

#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <unistd.h>

__device__
int getIndex(int x, int y, int rows){
	// (size * 3) * y + (x * 3)
	int result;
	result = (rows * 3) * y;
	result = result + (x * 3);
	return result;
}

__device__
int checkLeft(int index, int *gen, int rows)
{
	int indexNeighbour;

	int x = gen[index];
	int y = gen[index + 1];

	// 1
	if (x == 0)
	{
		indexNeighbour = getIndex((rows - 1), y, rows);
	} // 2
	else
	{
		indexNeighbour = getIndex((x - 1), y, rows);
		//indexNeighbour = index - 3;
	}

	if (gen[indexNeighbour + 2] == 1)
	{
		return 1;
	}
	else
	{
		return 0;
	}
}

__device__
int checkRight(int index, int *gen, int rows)
{
	int indexNeighbour;

	int x = gen[index];
	int y = gen[index + 1];

	// 1
	if (x == rows -1)
	{
		indexNeighbour = getIndex(0, y, rows);
	} // 2
	else
	{
		indexNeighbour = getIndex((x + 1), y, rows);
	}

	if (gen[indexNeighbour + 2] == 1)
	{
		return 1;
	}
	else
	{
		return 0;
	}
}

__device__
int checkTop(int index, int *gen, int rows, int columns)
{
	int indexNeighbour;

	int x = gen[index];
	int y = gen[index + 1];
	
	// 1
	if (y == 0)
	{
		indexNeighbour = getIndex(x , columns - 1, rows);
	} // 2
	else
	{
		indexNeighbour = getIndex(x , (y - 1), rows);
	}

	if (gen[indexNeighbour + 2] == 1)
	{
		return 1;
	}
	else
	{
		return 0;
	}
}

__device__
int checkBottom(int index, int *gen, int rows, int columns)
{
	int indexNeighbour;

	int x = gen[index];
	int y = gen[index + 1];;

	// 1
	if (y == columns - 1)
	{
		indexNeighbour = getIndex(x , 0 , rows);
	} // 2
	else
	{
		indexNeighbour = getIndex(x, (y + 1), rows);
	}
	
	if (gen[indexNeighbour + 2] == 1) 
	{
		return 1;
	}
	else
	{
		return 0;
	}
}

__device__
int checkDiagonalTL(int index, int *gen, int rows, int columns)
{
	int indexNeighbour;

	int x = gen[index];
	int y = gen[index + 1];

	int xNeighbour; 
	int yNeighbour;

	// 1
	if (x == 0 && y == 0)
	{
		xNeighbour = rows - 1;
		yNeighbour = columns - 1;
		indexNeighbour = getIndex(xNeighbour, yNeighbour, rows);
	}
	// 2
	else if (y == 0)
	{
		xNeighbour = x - 1;
		yNeighbour = columns - 1;
		indexNeighbour = getIndex(xNeighbour, yNeighbour, rows);
	}
	// 3
	else if (x == 0)
	{
		indexNeighbour = getIndex((rows -1), y - 1, rows);//index - 3;//getIndex((rows - 1), gen[index - 2));
	}
	// 4
	else
	{
		xNeighbour = x - 1;
		yNeighbour = y - 1;

		indexNeighbour = getIndex(xNeighbour, yNeighbour, rows);
	}

	if (gen[indexNeighbour + 2] == 1) //cells[y][xNeighbour].alive == 1)
	{
		return 1;
	}
	else
	{
		return 0;
	}
}

__device__
int checkDiagonalTR(int index, int *gen, int rows, int columns)
{
	int neighbourIndex;
	int x = gen[index]; 
	int y = gen[index + 1];
	int xNeighbour; 
	int yNeighbour;
	// 1
	if (x == (rows - 1) && y == 0)
	{
		xNeighbour = 0;
		yNeighbour = columns - 1;
		neighbourIndex = getIndex(xNeighbour, yNeighbour, rows);
	}
	// 2
	else if (y == 0)
	{
		xNeighbour = x + 1;
		yNeighbour = columns - 1;
		neighbourIndex = getIndex(xNeighbour, yNeighbour, rows);
	}
	// 3
	else if (x == (rows - 1))
	{
		xNeighbour = 0;
		yNeighbour = y - 1;
		neighbourIndex = getIndex(xNeighbour, yNeighbour, rows);
	}
	// 4
	else
	{
		xNeighbour = x + 1;
		yNeighbour = y -1;
		neighbourIndex = getIndex(xNeighbour, yNeighbour, rows);
	}

	if (gen[neighbourIndex + 2] == 1) //cells[y][xNeighbour].alive == 1)
	{
		return 1;
	}
	else
	{
		return 0;
	}
}

__device__
int checkDiagonalBL(int index, int *gen, int rows, int columns)
{
	int neighbourIndex;
	int x = gen[index]; 
	int y = gen[index + 1];
	int xNeighbour; 
	int yNeighbour;

	// 1
	if (x == 0 && y == (columns -1))
	{
		xNeighbour = rows - 1;
		yNeighbour = 0;
		neighbourIndex = getIndex(xNeighbour, yNeighbour, rows);
	}
	// 2
	else if (y == (columns - 1))
	{
		xNeighbour = x - 1;
		yNeighbour = 0;
		neighbourIndex = getIndex(xNeighbour, yNeighbour, rows);
	}
	// 3
	else if (x == 0)
	{
		xNeighbour = rows - 1;
		yNeighbour = y + 1;
		neighbourIndex = getIndex(xNeighbour, yNeighbour, rows);
	}
	// 4
	else
	{
		neighbourIndex = getIndex((x - 1), (y + 1), rows);
	}

	if (gen[neighbourIndex + 2] == 1) 
	{
		return 1;
	}
	else
	{
		return 0;
	}
}

__device__
int checkDiagonalBR(int index, int *gen, int rows, int columns)
{
	int neighbourIndex;
	int x = gen[index]; 
	int y = gen[index + 1];
	int xNeighbour; 
	int yNeighbour;

	// 1
	if (x == (rows - 1) && y == (columns -1))
	{
		xNeighbour = 0;
		yNeighbour = 0;
		neighbourIndex = 0;
	}
	// 2
	else if (y == (columns - 1))
	{
		xNeighbour = x + 1;
		yNeighbour = 0;
		neighbourIndex = getIndex(xNeighbour, yNeighbour, rows);
	}
	// 3
	else if (x == (rows - 1))
	{
		neighbourIndex = getIndex(0, (y + 1), rows);
	}
	// 4
	else
	{
		neighbourIndex = getIndex((x + 1), (y + 1), rows);
	}

	if (gen[neighbourIndex + 2] == 1) 
	{
		return 1;
	}
	else
	{
		return 0;
	}
}

__device__
int getCellNeighbours(int index, int *gen, int rows, int columns){
	int neighbours = 0;
	neighbours = checkLeft(index, gen, rows);
	neighbours = neighbours + checkRight(index, gen, rows);
	neighbours = neighbours + checkTop(index, gen, rows, columns);
	neighbours = neighbours + checkBottom(index, gen, rows, columns);
	neighbours = neighbours + checkDiagonalTL(index, gen, rows, columns);
	neighbours = neighbours + checkDiagonalTR(index, gen, rows, columns);
	neighbours = neighbours + checkDiagonalBL(index, gen, rows, columns);
	neighbours = neighbours + checkDiagonalBR(index, gen, rows, columns);
	
	return neighbours;
}

__device__
void cellNextCycle(int *gen, int *newGen, int index, int rows, int columns){

	int neighbours = 0;
	neighbours = getCellNeighbours(index, gen, rows, columns);
	
	// /printf("\nx : %d , y : %d. neighbours : %d", gen[index], gen[index + 1], neighbours);

	// Any live cell
	if (gen[index + 2] == 1)
	{
		//Any live cell with fewer than two live neighbours dies, as if caused by underpopulation.
		if (neighbours < 2)
		{
			newGen[index + 2] = 0;
		} //Any live cell with more than three live neighbours dies, as if by overpopulation.
		else if (neighbours == 2 || neighbours == 3)
		{
			newGen[index + 2] = 1;
		}
		else if (neighbours > 3)
		{
			newGen[index + 2] = 0;
		}
	}
	else if (neighbours == 3)
	{
		//printf("DEAD BECOMES ALIVE : x : %d , y : %d \n", gen[index], gen[index + 1]);
		//Any dead cell with exactly three live neighbours becomes a live cell, as if by reproduction.
		newGen[index + 2] = 1;
	}
	else{
		newGen[index + 2] = 0;
 	}
}

// Gets every cells next value which gets stored in newGen
__global__
void calculateBoard(int *gen, int *newGen, int amountofCells, int rows, int columns)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if(index >= amountofCells){
		return;
	}
	int count = index * 3;

	cellNextCycle(gen, newGen, count, rows, columns);
}

int getIndexCPU(int x, int y, int rows){
	// (size * 3) * y + (x * 3)
	int result;
	result = (rows * 3) * y;
	result = result + (x * 3);
	return result;
}

void print(int *gen, int amount){
	
	printf("\nPrinting Current Gen Array !! [x,y,alive] \n");
	printf("[\n");

	int count = 0;

	while(count < amount){
		printf("x : %d, y : %d, alive : %d \n", gen[count], gen[count + 1], gen[count + 2]);
		count = count + 3;
	}

	printf("]");
}

void printBoard(int *gen ,int amountofCells, int rows){

	int count = 0;
	int rowCount = 0;
	printf("\n");
	while(count < amountofCells){
		count++;
		count++;
		if(gen[count] == 0){
			printf(" . ");
		}else{
			printf(" x ");
		}
		rowCount++;

		if(rowCount == rows){
			printf("\n");
			rowCount = 0;
		}
		count++;
	}
}

void setupGlider(int *gen, int *newGen, int rows){

	int index;
	index = getIndexCPU(1,0,rows) + 2;
	gen[index] = 1;
	newGen[index] = 1;

	index = getIndexCPU(2,1,rows) + 2;
	gen[index] = 1;
	newGen[index] = 1;

	index = getIndexCPU(0,2,rows) + 2;
	gen[index] = 1;
	newGen[index] = 1;

	index = getIndexCPU(1,2,rows) + 2;
	gen[index] = 1;
	newGen[index] = 1;

	index = getIndexCPU(2,2,rows) + 2;
	gen[index] = 1;
	newGen[index] = 1;
}

void populateArray(int *gen, int rows, int columns){
	
	int count = 0;
    for (int y = 0; y < columns; y++) {
		for(int x = 0; x < rows; x++){
			gen[count] = x;
			count++;
			gen[count] = y;
			count++;
			gen[count] = 0;;
			count++;
		}
	}
}

int main(void){
	
	int *gen, *newGen;

	int rows = 50;
	int columns = 60;
	int amountofCells = rows * columns;
	int lengthofArray = ((amountofCells * 2) + amountofCells);
	int loopCount = 0;

	int amountOFBlocks = (amountofCells / 1024) + 1; 
	
	printf("User wants %d X %d , Total Cells needed : %d , Array Size : %d \n", rows , columns, amountofCells, lengthofArray);

	// Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&gen, lengthofArray*sizeof(int));
	hipMallocManaged(&newGen, lengthofArray*sizeof(int));

	// populate board
	populateArray(gen, rows, columns);
	populateArray(newGen, rows, columns);
	// set up glider
	setupGlider(gen, newGen, rows);

	hipDeviceSynchronize();

	// Keep calculating board & printing
	while(loopCount < 300){
		usleep(9000);
		if((loopCount % 2) == 0){
			calculateBoard<<<amountOFBlocks,1024>>>(gen, newGen, amountofCells, rows, columns);
			hipDeviceSynchronize();
			printBoard(newGen, lengthofArray, rows);
	
		} else{
			calculateBoard<<<amountOFBlocks,1024>>>(newGen, gen, amountofCells, rows, columns);
			hipDeviceSynchronize();
			printBoard(gen, lengthofArray, rows);
		}

		
		loopCount++;
	}

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	// Free memory
	hipFree(gen);
	hipFree(newGen);

	return 0;
}
	
