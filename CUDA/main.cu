// Amarjot Singh Parmar

#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <stdio.h>

__device__
int getIndex(int x, int y, int rows){
	// (size * 3) * y + (x * 3)
	int result;
	result = (rows * 3) * y;
	result = result + (x * 3);
	return result;
}

__device__
void printBoardd(int *gen ,int amountofCells, int rows){

	int count = 0;
	int rowCount = 0;
	printf("\n");
	while(count < amountofCells){
		count++;
		count++;
		if(gen[count] == 0){
			printf(" . ");
		}else{
			printf(" x ");
		}
		rowCount++;

		if(rowCount == rows){
			printf("\n");
			rowCount = 0;
		}
		count++;
	}
}
__device__
int checkLeft(int index, int *gen, int rows)
{
	int xNeighbour;
	int yNeighbour;
	int x = gen[index];
	int y = gen[index + 1];

	// 1
	if (gen[index] == 0)
	{
		xNeighbour = getIndex(rows - 1, y, rows);
	} // 2
	else
	{
		xNeighbour = index - 3;
	}

	if (gen[xNeighbour + 2] == 1)
	{
		return 1;
	}
	else
	{
		return 0;
	}
}

__device__
int checkRight(int index, int *gen, int rows)
{
	int xNeighbour;
	// 1
	if (gen[index] == rows -1)
	{
		xNeighbour = 0;
	} // 2
	else
	{
		xNeighbour = index + 3;
	}

	if (gen[xNeighbour + 2] == 1)
	{
		return 1;
	}
	else
	{
		return 0;
	}
}

__device__
int checkTop(int index, int *gen, int rows, int columns)
{
	int yNeighbour;
	// 1
	if (gen[index+1] == 0)
	{
		yNeighbour = getIndex(gen[index], columns - 1 , rows);
	} // 2
	else
	{
		yNeighbour = getIndex(gen[index], (gen[index + 1]- 1), rows);
	}

	if (gen[yNeighbour + 2] == 1)
	{
		return 1;
	}
	else
	{
		return 0;
	}
}

__device__
int checkBottom(int index, int *gen, int rows, int columns)
{
	int yNeighbour;

	// 1
	if (gen[index+1] == columns - 1)
	{
		yNeighbour = getIndex(gen[index], 0 , rows);
	} // 2
	else
	{
		yNeighbour = getIndex(gen[index], (gen[index + 1] + 1), rows);
	}
	
	if (gen[yNeighbour + 2] == 1) 
	{
		return 1;
	}
	else
	{
		return 0;
	}
}

__device__
int checkDiagonalTL(int index, int *gen, int rows, int columns)
{
	int neighbourIndex;
	int x = gen[index]; 
	int y = gen[index + 1];
	int xNeighbour; 
	int yNeighbour;

	// 1
	if (x == 0 && y == 0)
	{
		xNeighbour = rows - 1;
		yNeighbour = columns - 1;
		neighbourIndex = getIndex(xNeighbour, yNeighbour, rows);
	}
	// 2
	else if (y == 0)
	{
		xNeighbour = x - 1;
		yNeighbour = columns - 1;
		neighbourIndex = getIndex(xNeighbour, yNeighbour, rows);
	}
	// 3
	else if (x == 0)
	{
		neighbourIndex = index - 3;//getIndex((rows - 1), gen[index - 2));
	}
	// 4
	else
	{
		xNeighbour = x - 1;
		yNeighbour = y - 1;

		neighbourIndex = getIndex(xNeighbour, yNeighbour, rows);
	}

	if (gen[neighbourIndex + 2] == 1) //cells[y][xNeighbour].alive == 1)
	{
		return 1;
	}
	else
	{
		return 0;
	}
}

__device__
int checkDiagonalTR(int index, int *gen, int rows, int columns)
{
	int neighbourIndex;
	int x = gen[index]; 
	int y = gen[index + 1];
	int xNeighbour; 
	int yNeighbour;
	// 1
	if (x == (rows - 1) && y == 0)
	{
		xNeighbour = 0;
		yNeighbour = columns - 1;
		neighbourIndex = getIndex(xNeighbour, yNeighbour, rows);
	}
	// 2
	else if (y == 0)
	{
		xNeighbour = x + 1;
		yNeighbour = columns - 1;
		neighbourIndex = getIndex(xNeighbour, yNeighbour, rows);
	}
	// 3
	else if (x == (rows - 1))
	{
		xNeighbour = 0;
		yNeighbour = y - 1;
		neighbourIndex = getIndex(xNeighbour, yNeighbour, rows);
	}
	// 4
	else
	{
		xNeighbour = x + 1;
		yNeighbour = y -1;
		neighbourIndex = getIndex(xNeighbour, yNeighbour, rows);
	}

	if (gen[neighbourIndex + 2] == 1) //cells[y][xNeighbour].alive == 1)
	{
		return 1;
	}
	else
	{
		return 0;
	}
}

__device__
int checkDiagonalBL(int index, int *gen, int rows, int columns)
{
	int neighbourIndex;
	int x = gen[index]; 
	int y = gen[index + 1];
	int xNeighbour; 
	int yNeighbour;
	// 1
	if (x == 0 && y == (columns -1))
	{
		xNeighbour = rows - 1;
		yNeighbour = 0;
		neighbourIndex = getIndex(x, y, rows);
	}
	// 2
	else if (y == (columns - 1))
	{
		xNeighbour = x - 1;
		yNeighbour = 0;
		neighbourIndex = getIndex(xNeighbour, yNeighbour, rows);
	}
	// 3
	else if (x == 0)
	{
		xNeighbour = rows - 1;
		yNeighbour = y + 1;
		neighbourIndex = getIndex(xNeighbour, yNeighbour, rows);
	}
	// 4
	else
	{
		neighbourIndex = getIndex(x - 1, y + 1, rows);
	}

	if (gen[neighbourIndex + 2] == 1) 
	{
		return 1;
	}
	else
	{
		return 0;
	}
}

__device__
int checkDiagonalBR(int index, int *gen, int rows, int columns)
{
	int neighbourIndex;
	int x = gen[index]; 
	int y = gen[index + 1];
	int xNeighbour; 
	int yNeighbour;
	// 1
	if (x == (rows - 1) && y == (columns -1))
	{
		xNeighbour = 0;
		yNeighbour = 0;
		neighbourIndex = getIndex(x, y, rows);
	}
	// 2
	else if (y == (columns - 1))
	{
		xNeighbour = x + 1;
		yNeighbour = 0;
		neighbourIndex = getIndex(xNeighbour, yNeighbour, rows);
	}
	// 3
	else if (x == (rows - 1))
	{
		neighbourIndex = index + 3;
	}
	// 4
	else
	{
		neighbourIndex = getIndex(x + 1, y + 1, rows);
	}

	if (gen[neighbourIndex + 2] == 1) 
	{
		return 1;
	}
	else
	{
		return 0;
	}
}

__device__
int getCellNeighbours(int index, int *gen, int rows, int columns){
	int neighbours = 0;
	neighbours = checkLeft(index, gen, rows);
	neighbours = neighbours + checkRight(index, gen, rows);
	neighbours = neighbours + checkTop(index, gen, rows, columns);
	neighbours = neighbours + checkBottom(index, gen, rows, columns);
	neighbours = neighbours + checkDiagonalTL(index, gen, rows, columns);
	neighbours = neighbours + checkDiagonalTR(index, gen, rows, columns);
	neighbours = neighbours + checkDiagonalBL(index, gen, rows, columns);
	neighbours = neighbours + checkDiagonalBR(index, gen, rows, columns);
	/*
	printf(" x : %d , y : %d. Check LEft neighbours : %d \n", gen[index], gen[index + 1], neighbours);
	printf(" x : %d , y : %d. Check right neighbours : %d \n", gen[index], gen[index + 1], checkRight(index, gen, rows));
	printf(" x : %d , y : %d. Check Top neighbours : %d \n", gen[index], gen[index + 1], checkTop(index, gen, rows, columns));
	printf(" x : %d , y : %d. Check B neighbours : %d \n", gen[index], gen[index + 1], checkBottom(index, gen, rows, columns));
	printf(" x : %d , y : %d. Check TL neighbours : %d \n", gen[index], gen[index + 1], checkDiagonalTL(index, gen, rows, columns));
	printf(" x : %d , y : %d. Check TR neighbours : %d \n", gen[index], gen[index + 1], checkDiagonalTR(index, gen, rows, columns));
	printf(" x : %d , y : %d. Check BL neighbours : %d \n", gen[index], gen[index + 1], checkDiagonalBL(index, gen, rows, columns));
	printf(" x : %d , y : %d. Check BR neighbours : %d \n", gen[index], gen[index + 1], checkDiagonalBR(index, gen, rows, columns));
	*/
	return neighbours;
}

__device__
void cellNextCycle(int *gen, int *newGen, int index, int rows, int columns){

	int neighbours = 0;
	neighbours = getCellNeighbours(index, gen, rows, columns);
	
	// /printf("\nx : %d , y : %d. neighbours : %d", gen[index], gen[index + 1], neighbours);

	// Any live cell
	if (gen[index + 2] == 1)
	{
		//Any live cell with fewer than two live neighbours dies, as if caused by underpopulation.
		if (neighbours < 2)
		{
			newGen[index + 2] = 0;
		} //Any live cell with more than three live neighbours dies, as if by overpopulation.
		else if (neighbours == 2 || neighbours == 3)
		{
			newGen[index + 2] = 1;
		}
		else if (neighbours > 3)
		{
			newGen[index + 2] = 0;
		}
	}
	else
	{
		//Any dead cell with exactly three live neighbours becomes a live cell, as if by reproduction.
		if (neighbours == 3 && gen[index + 2] == 0)
		{
			newGen[index + 2] = 1;
		}
	}
}

// Gets every cells next value which gets stored in newGen
__global__
void calculateBoard(int *gen, int *newGen, int amountofCells, int switchh, int rows, int columns)
{
	int count = 0;

	for(int x = 0; x < amountofCells; x++){
		cellNextCycle(gen, newGen, count, rows, columns);
		count = count + 3;
	}
	

	for(int x = 0; x < ((amountofCells * 3) -1); x++){
		gen[x] = newGen[x];
	}	
}

int getIndexCPU(int x, int y, int rows){
	// (size * 3) * y + (x * 3)
	int result;
	result = (rows * 3) * y;
	result = result + (x * 3);
	return result;
}

void print(int *gen, int amount){
	
	printf("\nPrinting Current Gen Array !! [x,y,alive] \n");
	printf("[\n");

	int count = 0;

	while(count < amount){
		printf("x : %d, y : %d, alive : %d \n", gen[count], gen[count + 1], gen[count + 2]);
		count = count + 3;
	}

	printf("]");
}

void printBoard(int *gen ,int amountofCells, int rows){

	int count = 0;
	int rowCount = 0;
	printf("\n");
	while(count < amountofCells){
		count++;
		count++;
		if(gen[count] == 0){
			printf(" . ");
		}else{
			printf(" x ");
		}
		rowCount++;

		if(rowCount == rows){
			printf("\n");
			rowCount = 0;
		}
		count++;
	}
}

void setupGlider(int *gen, int *newGen, int rows){

	int index;
	index = getIndexCPU(1,0,rows) + 2;
	gen[index] = 1;
	newGen[index] = 1;

	index = getIndexCPU(2,1,rows) + 2;
	gen[index] = 1;
	newGen[index] = 1;

	index = getIndexCPU(0,2,rows) + 2;
	gen[index] = 1;
	newGen[index] = 1;

	index = getIndexCPU(1,2,rows) + 2;
	gen[index] = 1;
	newGen[index] = 1;

	index = getIndexCPU(2,2,rows) + 2;
	gen[index] = 1;
	newGen[index] = 1;
}

void populateArrays(int *gen, int *newGen, int rows, int columns){
	
	int count = 0;
    for (int y = 0; y < columns; y++) {
		for(int x = 0; x < rows; x++){
			gen[count] = x;
			newGen[count] = x;
			count++;
			gen[count] = y;
			newGen[count] = y;
			count++;
			gen[count] = 0;
			newGen[count] = 0;
			count++;
		}
	}
}

int main(void){
	
	int *gen, *newGen;

	int rows = 50;
	int columns = 50;
	int amountofCells = rows * columns;
	int lengthofArray = ((amountofCells * 2) + amountofCells);
	int loopCount = 0;
	
	printf("User wants %d X %d , Total Cells needed : %d , Array Size : %d \n", rows , columns, amountofCells, lengthofArray);

	// Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&gen, lengthofArray*sizeof(int));
	hipMallocManaged(&newGen, lengthofArray*sizeof(int));

	// populate board
	populateArrays(gen, newGen, rows, columns);

	// set up glider
	setupGlider(gen, newGen, rows);
	hipDeviceSynchronize();

	// Keep calculating board & printing
	while(loopCount < 30){

		calculateBoard<<<1,1>>>(gen, newGen, amountofCells, loopCount, rows, columns);
		hipDeviceSynchronize();
		printBoard(gen, lengthofArray, rows);
		
		loopCount++;
	}

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	// Free memory
	hipFree(gen);
	hipFree(newGen);

	return 0;
}
	
