// Amarjot Singh Parmar

#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <unistd.h>

__device__
int getIndex(int x, int y, int rows){
	// (size * 3) * y + (x * 3)
	int result = (rows * 3) * y;
	result = result + (x * 3);
	return result;
}

__device__
int getCellNeighbours(int index, int *gen, int rows, int columns){
	int neighbours = 0;
	int indexNeighbour, xNeighbour, yNeighbour;
	int x = gen[index];
	int y = gen[index + 1];
	// L
	if (x == 0){
		indexNeighbour = getIndex((rows - 1), y, rows);
	} 
	else{
		indexNeighbour = getIndex((x - 1), y, rows);
	}
	neighbours = gen[indexNeighbour + 2];
	// R
	if (x == rows -1){
		indexNeighbour = getIndex(0, y, rows);
	} 
	else{
		indexNeighbour = getIndex((x + 1), y, rows);
	}
	neighbours = neighbours + gen[indexNeighbour + 2];
	// T
	if (y == 0){
		indexNeighbour = getIndex(x , columns - 1, rows);
	}
	else{
		indexNeighbour = getIndex(x , (y - 1), rows);
	}
	neighbours = neighbours + gen[indexNeighbour + 2];
	// N
	if (y == columns - 1){
		indexNeighbour = getIndex(x , 0 , rows);
	} 
	else{
		indexNeighbour = getIndex(x, (y + 1), rows);
	}
	neighbours = neighbours + gen[indexNeighbour + 2];
	// TL
	if (x == 0 && y == 0){
		xNeighbour = rows - 1;
		yNeighbour = columns - 1;
		indexNeighbour = getIndex(xNeighbour, yNeighbour, rows);
	} 
	else if (y == 0){
		xNeighbour = x - 1;
		yNeighbour = columns - 1;
		indexNeighbour = getIndex(xNeighbour, yNeighbour, rows);
	} 
	else if (x == 0){
		indexNeighbour = getIndex((rows -1), y - 1, rows);
	} 
	else{
		xNeighbour = x - 1;
		yNeighbour = y - 1;

		indexNeighbour = getIndex(xNeighbour, yNeighbour, rows);
	}
	neighbours = neighbours + gen[indexNeighbour + 2];
	// TR
	if (x == (rows - 1) && y == 0){
		xNeighbour = 0;
		yNeighbour = columns - 1;
		indexNeighbour = getIndex(xNeighbour, yNeighbour, rows);
	}
	else if (y == 0){
		xNeighbour = x + 1;
		yNeighbour = columns - 1;
		indexNeighbour = getIndex(xNeighbour, yNeighbour, rows);
	}
	else if (x == (rows - 1)){
		xNeighbour = 0;
		yNeighbour = y - 1;
		indexNeighbour = getIndex(xNeighbour, yNeighbour, rows);
	}
	else{
		xNeighbour = x + 1;
		yNeighbour = y -1;
		indexNeighbour = getIndex(xNeighbour, yNeighbour, rows);
	}
	neighbours = neighbours + gen[indexNeighbour + 2];
	// BL
	if (x == 0 && y == (columns -1)){
		xNeighbour = rows - 1;
		yNeighbour = 0;
		indexNeighbour = getIndex(xNeighbour, yNeighbour, rows);
	}
	else if (y == (columns - 1)){
		xNeighbour = x - 1;
		yNeighbour = 0;
		indexNeighbour = getIndex(xNeighbour, yNeighbour, rows);
	}
	else if (x == 0){
		xNeighbour = rows - 1;
		yNeighbour = y + 1;
		indexNeighbour = getIndex(xNeighbour, yNeighbour, rows);
	}
	else{
		indexNeighbour = getIndex((x - 1), (y + 1), rows);
	}
	neighbours = neighbours + gen[indexNeighbour + 2];
	// BR
	if (x == (rows - 1) && y == (columns -1)){
		xNeighbour = 0;
		yNeighbour = 0;
		indexNeighbour = 0;
	}
	else if (y == (columns - 1)){
		xNeighbour = x + 1;
		yNeighbour = 0;
		indexNeighbour = getIndex(xNeighbour, yNeighbour, rows);
	}
	else if (x == (rows - 1)){
		indexNeighbour = getIndex(0, (y + 1), rows);
	}
	else{
		indexNeighbour = getIndex((x + 1), (y + 1), rows);
	}
	neighbours = neighbours + gen[indexNeighbour + 2];
	
	return neighbours;
}

__device__
void cellNextCycle(int *gen, int *newGen, int index, int rows, int columns){

	int neighbours = getCellNeighbours(index, gen, rows, columns);
	// Any live cell
	if (gen[index + 2] == 1){
		//Any live cell with fewer than two live neighbours dies, as if caused by underpopulation.
		if (neighbours < 2)
		{
			newGen[index + 2] = 0;
		} //Any live cell with more than three live neighbours dies, as if by overpopulation.
		else if (neighbours == 2 || neighbours == 3){
			newGen[index + 2] = 1;
		}
		else if (neighbours > 3){
			newGen[index + 2] = 0;
		}
	}
	else if (neighbours == 3){
		//Any dead cell with exactly three live neighbours becomes a live cell, as if by reproduction.
		newGen[index + 2] = 1;
	}
	else{
		newGen[index + 2] = 0;
 	}
}

// Gets every cells next value which gets stored in newGen
__global__
void calculateBoard(int *gen, int *newGen, int amountofCells, int rows, int columns){
	
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if(index >= amountofCells){
		return;
	}
	int count = index * 3;

	cellNextCycle(gen, newGen, count, rows, columns);
}

int getIndexCPU(int x, int y, int rows){
	// (size * 3) * y + (x * 3)
	int result;
	result = (rows * 3) * y;
	result = result + (x * 3);
	return result;
}

void printBoard(int *gen ,int amountofCells, int rows){
	int count = 0;
	int rowCount = 0;
	printf("\n");
	while(count < amountofCells){
		count++;
		count++;
		if(gen[count] == 0){
			printf(" . ");
		}else{
			printf(" x ");
		}
		rowCount++;

		if(rowCount == rows){
			printf("\n");
			rowCount = 0;
		}
		count++;
	}
}

void setupGlider(int *gen, int *newGen, int rows){

	int index;
	index = getIndexCPU(1,0,rows) + 2;
	gen[index] = 1;

	index = getIndexCPU(2,1,rows) + 2;
	gen[index] = 1;

	index = getIndexCPU(0,2,rows) + 2;
	gen[index] = 1;

	index = getIndexCPU(1,2,rows) + 2;
	gen[index] = 1;

	index = getIndexCPU(2,2,rows) + 2;
	gen[index] = 1;
}

void populateArray(int *gen, int rows, int columns){

	int count = 0;
    for (int y = 0; y < columns; y++) {
		for(int x = 0; x < rows; x++){
			gen[count] = x;
			count++;
			gen[count] = y;
			count++;
			gen[count] = 0;;
			count++;
		}
	}
}

int main(void){

	int rows = 50;
	int columns = 50;
	int runTime = 1000;

	int amountofCells = rows * columns;
	int lengthofArray = ((amountofCells * 2) + amountofCells);
	int loopCount = 0;
	int amountOFBlocks = (amountofCells / 1024) + 1; 
	
	printf("User wants %d X %d , Total Cells needed : %d , Array Size : %d \n", rows , columns, amountofCells, lengthofArray);

	// Allocate Unified Memory – accessible from CPU or GPU
	int *gen, *newGen;
    hipMallocManaged(&gen, lengthofArray*sizeof(int));
	hipMallocManaged(&newGen, lengthofArray*sizeof(int));

	// populate board
	populateArray(gen, rows, columns);
	populateArray(newGen, rows, columns);
	setupGlider(gen, newGen, rows);

	hipDeviceSynchronize();

	// Keep calculating board & printing
	while(loopCount < runTime){
		usleep(9000);
		if((loopCount % 2) == 0){
			calculateBoard<<<amountOFBlocks,1024>>>(gen, newGen, amountofCells, rows, columns);
			hipDeviceSynchronize();
			printBoard(newGen, lengthofArray, rows);
		} else{
			calculateBoard<<<amountOFBlocks,1024>>>(newGen, gen, amountofCells, rows, columns);
			hipDeviceSynchronize();
			printBoard(gen, lengthofArray, rows);
		}
		loopCount++;
	}
	hipFree(gen);
	hipFree(newGen);

	return 0;
}