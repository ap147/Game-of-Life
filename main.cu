
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <stdio.h>

__device__ float *a;

// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
	  int index = threadIdx.x;
	  int stride = blockDim.x;

  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}

__global__
void print(int size, int *x, int *y, int *alive)
{
	  int index = threadIdx.x;
	  int stride = blockDim.x;

	  for (int i = index; i < size; i += stride)
	  printf("COUNT : %d -- x : %d, y : %d , alive : %d \n", x[i], y[i], alive[i]);
}

__global__
void printBoard(int size, int *alive)
{
	int count = 0;
	// initialize x and y arrays on the host
	for (int y = 0; y < size; y++) {
		for(int x = 0; x < size; x++){
			if(alive[count] == 1){
				printf(" x ");
			}
			else{
				printf(" . ");
			}
			count++;
		}
		printf(" \n");
	}
}

__global__
void getIndex(int x, int y, int size){
	int result;
	result = y * size;
	result = result + x;
}

int main(void)
{
  int *xPos, *yPos, *alive, *nextState;
  int size = 30;
  int amount = size * size;
  int count = 0;


  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&xPos, amount*sizeof(int));
  hipMallocManaged(&yPos, amount*sizeof(int));
  hipMallocManaged(&alive, amount*sizeof(int));
  hipMallocManaged(&nextState, amount*sizeof(int));

  // initialize x and y arrays on the host
  for (int y = 0; y < size; y++) {
	  for(int x = 0; x < size; x++){
		  xPos[count] = x;
		  yPos[count] = y;
		  alive[count] = 0;
		  nextState[count] = 0;
		  count++;
	  }
  }

  // Get GPU to do this.
  int result;
	result = 0 * size;
	result = result + 1;
	alive[result] = 1;

	result = 1 * size;
	result = result + 2;
	alive[result] = 1;

	result = 2 * size;
	result = result + 0;
	alive[result] = 1;

	result = 2 * size;
	result = result + 1;
	alive[result] = 1;

	result = 2 * size;
	result = result + 2;
	alive[result] = 1;

  printBoard<<<1, 1>>>(size, alive);

  // setupGlider<<<1,1>>>(xPos, yPos, alive);
  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Free memory
  hipFree(xPos);
  hipFree(yPos);
  hipFree(alive);
  hipFree(nextState);

  return 0;
}

