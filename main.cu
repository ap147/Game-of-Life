// Amarjot Singh Parmar

#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <stdio.h>


__global__
void print(int *gen, int amount){
	
	printf("\nPrinting Current Gen Array !! [x,y,alive] \n");
	printf("[\n");

	int count = 0;

	while(count < amount){
		printf("x : %d, y : %d, alive : %d \n", gen[count], gen[count + 1], gen[count + 2]);
		count = count + 3;
	}

	printf("]");
}

// I can do this in parallel right ?
__global__
void printBoard(int *gen ,int amountofCells, int size){

	int count = 0;
	int rowCount = 0;
	printf("\n");
	while(count < amountofCells){
		count++;
		count++;
		if(gen[count] == 0){
			printf(" . ");
		}else{
			printf(" x ");
		}
		rowCount++;

		if(rowCount == size){
			printf("\n");
			rowCount = 0;
		}

		count++;
		
	}
}

// I can do this in parallel right ?
__global__
void populateArrays(int *gen, int *newGen, int size){
	
	int count = 0;
    for (int y = 0; y < size; y++) {
		for(int x = 0; x < size; x++){

			gen[count] = x;
			newGen[count] = x;
			count++;
			gen[count] = y;
			newGen[count] = y;
			count++;
			gen[count] = 0;
			newGen[count] = 0;
			count++;
		}
	}
}

__global__
void getIndex(int x, int y, int size){
	int result;
	result = y * size;
	result = result + x;
}

__global__
void calculateCells(int *gen, int *newGen){

}

int main(void){
	
	int *gen, *newGen;
	int size = 5;
	int amountofCells = size * size;
	int lengthofArray = ((amountofCells * 2) + amountofCells);
	int loopCount = 0;
	
	printf("User wants size : %d , Total Cells needed : %d , Array Size : %d \n", size, (size * size), lengthofArray);

	// Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&gen, lengthofArray*sizeof(int));
	hipMallocManaged(&newGen, lengthofArray*sizeof(int));

	// populate board
	populateArrays<<<1,1>>>(gen, newGen, size);

	// setting up glider

	// Keep calculating board & printing

	while(loopCount < 1){
		printBoard<<<1,1>>>(gen, lengthofArray, size);
		loopCount++;
	}
	
	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();
	// Free memory
	hipFree(gen);
	hipFree(newGen);
	return 0;
}

