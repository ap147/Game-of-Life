
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <stdio.h>

__device__ float *a;

// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  for (int i = 0; i < n; i++)
    y[i] = x[i] + y[i];
}

__global__
void print(int size, int *x, int *y, int *alive)
{
  for (int i = 0; i < 9; i++)
	  printf("x : %d, y : %d , alive : %d \n", x[i], y[i], alive[i]);
}

__global__
void printBoard(int size, int *alive)
{
	int count = 0;
	// initialize x and y arrays on the host
	for (int y = 0; y < size; y++) {
		for(int x = 0; x < size; x++){
			if(alive[count] == 1){
				printf(" x ");
			}
			else{
				printf(" . ");
			}
			count++;
		}
		printf(" \n");
	}
}

__global__
void getIndex(int x, int y, int size){
	int result;
	result = y * size;
	result = result + x;
}

int main(void)
{
  int *xPos, *yPos, *alive, *nextState;
  int size = 3;
  int count = 0<<0;
  const size_t sz = 10 * sizeof(float);
  float *ah;
     hipMalloc((void **)&ah, sz);
     hipMemcpyToSymbol(HIP_SYMBOL("a"), &ah, sizeof(float *), size_t(0),hipMemcpyHostToDevice);

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&xPos, size*sizeof(int));
  hipMallocManaged(&yPos, size*sizeof(int));
  hipMallocManaged(&alive, size*sizeof(int));
  hipMallocManaged(&nextState, size*sizeof(int));

  // initialize x and y arrays on the host
  for (int y = 0; y < size; y++) {
	  for(int x = 0; x < size; x++){
		  xPos[count] = x;
		  yPos[count] = y;
		  alive[count] = 0;
		  nextState[count] = 0;
		  count++;
	  }
  }


  // Run kernel on 1M elements on the GPU
  print<<<1, 1>>>(size, xPos, yPos, alive);


	int result;
	result = 0 * size;
	result = result + 1;
	alive[result] = 1;

	result = 1 * size;
	result = result + 2;
	alive[result] = 1;

	result = 2 * size;
	result = result + 0;
	alive[result] = 1;

	result = 2 * size;
	result = result + 1;
	alive[result] = 1;

	result = 2 * size;
	result = result + 2;
	alive[result] = 1;
/*
 *
  getIndex<<<1,1>>>(1, 0, size, result);
  alive[result] = 1;
  getIndex<<<1,1>>>(2, 1, size, result);
  printf("result : %d \n",result);
  alive[result] = 1;
  getIndex<<<1,1>>>(0, 2, size, result);
  alive[result] = 1;
  getIndex<<<1,1>>>(1, 2, size, result);
  alive[result] = 1;
  getIndex<<<1,1>>>(2, 2, size, result);
  alive[result] = 1;

  print<<<1, 1>>>(size, xPos, yPos, alive);
*/
	hipDeviceSynchronize();
  printBoard<<<1, 1>>>(size, alive);
  // setupGlider<<<1,1>>>(xPos, yPos, alive);
  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();


  // Free memory
  hipFree(xPos);
  hipFree(yPos);
  hipFree(alive);
  hipFree(nextState);

  return 0;
}
