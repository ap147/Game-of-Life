// Amarjot Singh Parmar

#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <stdio.h>


__global__
void print(int *gen, int amount){
	
	printf("\nPrinting Current Gen Array !! [x,y,alive] \n");
	printf("[\n");

	int count = 0;

	while(count < amount){
		printf("x : %d, y : %d, alive : %d \n", gen[count], gen[count + 1], gen[count + 2]);
		count = count + 3;
	}

	printf("]");
}

// I can do this in parallel right ?
__global__
void printBoard(int *gen ,int amountofCells, int size){

	int count = 0;
	int rowCount = 0;
	printf("\n");
	while(count < amountofCells){
		count++;
		count++;
		if(gen[count] == 0){
			printf(" . ");
		}else{
			printf(" x ");
		}
		rowCount++;

		if(rowCount == size){
			printf("\n");
			rowCount = 0;
		}
		count++;
	}
}

// I can do this in parallel right ?
__global__
void populateArrays(int *gen, int *newGen, int size){
	
	int count = 0;
    for (int y = 0; y < size; y++) {
		for(int x = 0; x < size; x++){

			gen[count] = x;
			newGen[count] = x;
			count++;
			gen[count] = y;
			newGen[count] = y;
			count++;
			gen[count] = 0;
			newGen[count] = 0;
			count++;
		}
	}
}

__device__
int getIndex(int x, int y, int size){
	// (size * 3) * y + (x * 3)
	int result;
	result = (size * 3) * y;
	result = result + (x * 3);
	return result;
}

__global__ 
void setupGlider(int *gen, int size){

	int index;
	index = getIndex(1,0,size) + 2;
	gen[index] = 1;

	index = getIndex(2,1,size) + 2;
	gen[index] = 1;

	index = getIndex(0,2,size) + 2;
	gen[index] = 1;

	index = getIndex(1,2,size) + 2;
	gen[index] = 1;

	index = getIndex(2,2,size) + 2;
	gen[index] = 1;
}


__device__
int getCellNeighbours(int xPos, int yPos){
	return 0;
}
__device__
void cellNextCycle(int *gen, int *newGen, int n){
	int neighbours = 0;

	  // Any live cell
	if (gen[2]== 1)
	{
		//Any live cell with fewer than two live neighbours dies, as if caused by underpopulation.
		if (neighbours < 2)
		{
			//cells[y][x].nextState = 0;
		} //Any live cell with more than three live neighbours dies, as if by overpopulation.
		else if (neighbours == 2 || neighbours == 3)
		{
			//cells[y][x].nextState = 1;
		}
		else if (neighbours > 3)
		{
			//cells[y][x].nextState = 0;
		}
	}
	else
	{
		//Any dead cell with exactly three live neighbours becomes a live cell, as if by reproduction.
		if (neighbours == 3 && gen[3] == 0)
		{
			//cells[y][x].nextState = 1;
		}
	}
}

// Gets every cells next value which gets stored in newGen
// 
__global__
void calculateBoard(int *gen, int *newGen){





}

int main(void){
	
	int *gen, *newGen;
	int size = 5;
	int amountofCells = size * size;
	int lengthofArray = ((amountofCells * 2) + amountofCells);
	int loopCount = 0;
	
	printf("User wants size : %d , Total Cells needed : %d , Array Size : %d \n", size, (size * size), lengthofArray);

	// Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&gen, lengthofArray*sizeof(int));
	hipMallocManaged(&newGen, lengthofArray*sizeof(int));

	// populate board
	populateArrays<<<1,1>>>(gen, newGen, size);

	// set up glider
	setupGlider<<<1,1>>>(gen, size);

	// Keep calculating board & printing
	while(loopCount < 1){
		printBoard<<<1,1>>>(gen, lengthofArray, size);
		//calculateBoard<<1,amountofCells>>>(gen, newGen);
		loopCount++;
	}
	
	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();
	// Free memory
	hipFree(gen);
	hipFree(newGen);
	return 0;
}

